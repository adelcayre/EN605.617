#include "hip/hip_runtime.h"
#include <stdio.h>
    
//array size
const int N=2048;
    

//add shared
__global__ void Add(int* Arr_A, int* Arr_B, int* Arr_C)
{
   
   __shared__ int a[N];
   __shared__ int b[N];
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   a[i]=Arr_A[i];
   b[i]=Arr_B[i];
   __syncthreads();
   if(i<N) Arr_C[i] = a[i] + b[i];
                
    
}

//subtract shared
__global__ void Subtract(int* Arr_A, int* Arr_B, int* Arr_C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int a[N];
    __shared__ int b[N];
    a[i]=Arr_A[i];
    b[i]=Arr_B[i];
   __syncthreads();
    if(i<N) Arr_C[i] = a[i] - b[i];
    
} 

//mult shared
__global__ void Mult(int* Arr_A, int* Arr_B, int* Arr_C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int a[N];
    __shared__ int b[N];
    a[i]=Arr_A[i];
    b[i]=Arr_B[i];
   __syncthreads();
    if(i<N) Arr_C[i] = a[i] * b[i];
    
} 

//mod shared
__global__ void Mod(int* Arr_A, int* Arr_B, int* Arr_C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int a[N];
    __shared__ int b[N];
    a[i]=Arr_A[i];
    b[i]=Arr_B[i];
   __syncthreads();
    if(i<N) Arr_C[i] = a[i] % b[i];
    
} 
    

//function for launching kernels with shared memory                           
__host__ void launchKernelShared(char op[], int blocks, int threads, int blockSize){
     
    //declare pointers
    int *a, *b, *c, *dev_a, *dev_b, *dev_c;
    
    //HOST pinned memory allocation
    hipHostMalloc((void **)&a, N*sizeof(int));
    hipHostMalloc((void **)&b, N*sizeof(int));
    hipHostMalloc((void **)&c, N*sizeof(int));
    
    //GPU memory allocation
    hipMalloc(&dev_a, N * sizeof(int));
    hipMalloc(&dev_b, N * sizeof(int));
    hipMalloc(&dev_c, N * sizeof(int));
    
    //populate host arrays
    srand ( time(NULL) );
    for (int i = 0; i < N; i++) {
        a[i] = i; //a contains index number
        b[i] = rand() % 4; //b contains randoom values 0 to 3
    }
    
    //copy host arrays to GPU
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    
                          
    //performace measurement events
    hipEvent_t kernel_start, kernel_stop;
    hipEventCreate(&kernel_start,0);
    hipEventCreate(&kernel_stop,0);
                          
    //select kernel to launch
    if(strcmp(op, "add")==0){
        hipEventRecord(kernel_start, 0);
        Add<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    }
    
    else if(strcmp(op, "subtract")==0){
        hipEventRecord(kernel_start, 0);
        Subtract<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    }
    
    else if(strcmp(op, "multiply")==0){
        hipEventRecord(kernel_start, 0);
        Mult<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    }
    
    else if(strcmp(op, "mod")==0){
        hipEventRecord(kernel_start, 0);
        Mod<<<blocks, threads>>>(dev_a, dev_b, dev_c);
        
    }
    
    
    
    
    //copy result back to host
    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
    
    //synchronize 
    hipEventRecord(kernel_stop, 0);
    hipEventSynchronize(kernel_stop);
    hipDeviceSynchronize();
    
    //print first 10 elements of result
    for (int i = 0; i < 10; i++) {
        printf("%d %d %d \n", a[i], b[i], c[i]);
    }
    
    
    float elapsedTime =0.0F;
    hipEventElapsedTime(&elapsedTime, kernel_start, kernel_stop);
    printf("Processed %d %s operations with %d threads and %d blocks (%d threads per block) in %f seconds using shared memmory \n", N, op, threads, blocks, blockSize, elapsedTime);
   
    
    //free allocated memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    
    //destroy events
    hipEventDestroy(kernel_start);
    hipEventDestroy(kernel_stop);
    
    
} 

//function for launching kernels with shared memory with streams                          
__host__ void launchKernelSharedStream(char op[], int blocks, int threads, int blockSize){
     
    //declare pointers
    int *a, *b, *c, *dev_a, *dev_b, *dev_c;
    
    //declare stream
    hipStream_t stream; 
    hipStreamCreate(&stream);
    
    
    //HOST pinned memory allocation
    hipHostMalloc((void **)&a, N*sizeof(int));
    hipHostMalloc((void **)&b, N*sizeof(int));
    hipHostMalloc((void **)&c, N*sizeof(int));
    
    //GPU memory allocation
    hipMalloc(&dev_a, N * sizeof(int));
    hipMalloc(&dev_b, N * sizeof(int));
    hipMalloc(&dev_c, N * sizeof(int));
    
    //populate host arrays
    srand ( time(NULL) );
    for (int i = 0; i < N; i++) {
        a[i] = i; //a contains index number
        b[i] = rand() % 4; //b contains randoom values 0 to 3
    }
    
    //copy host arrays to GPU
    hipMemcpyAsync(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice, stream);
    
                          
    //performace measurement events
    hipEvent_t kernel_start, kernel_stop;
    hipEventCreate(&kernel_start,0);
    hipEventCreate(&kernel_stop,0);
    
                          
    //select kernel to launch
    if(strcmp(op, "add")==0){
        hipEventRecord(kernel_start, 0);
        Add<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    }
    
    else if(strcmp(op, "subtract")==0){
        hipEventRecord(kernel_start, 0);
        Subtract<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    }
    
    else if(strcmp(op, "multiply")==0){
        hipEventRecord(kernel_start, 0);
        Mult<<<blocks, threads>>>(dev_a, dev_b, dev_c);
        
    }
    
    else if(strcmp(op, "mod")==0){
        hipEventRecord(kernel_start, 0);
        Mod<<<blocks, threads>>>(dev_a, dev_b, dev_c);
        
    }
    
    
    
    //copy result back to host
    hipMemcpyAsync(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost, stream);
    
    //synchronize
    hipStreamSynchronize(stream);
    hipEventRecord(kernel_stop, 0);
    hipEventSynchronize(kernel_stop);
    
    
    
    
    //print first 10 elements of result
    for (int i = 0; i < 10; i++) {
        printf("%d %d %d \n", a[i], b[i], c[i]);
    }
    
    
    float elapsedTime =0.0F;
    hipEventElapsedTime(&elapsedTime, kernel_start, kernel_stop);
    printf("Processed %d %s operations with %d threads and %d blocks (%d threads per block) in %f seconds using shared memmory \n", N, op, threads, blocks, blockSize, elapsedTime);
   
    
    //free allocated memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    
    //destroy events and stream
    hipEventDestroy(kernel_start);
    hipEventDestroy(kernel_stop);
    hipStreamDestroy(stream);
    
    
}     
                           
//launches a kernel without output to "warm up" GPU??????                          
__host__ void launchKernelVoid(char op[], int blocks, int threads, int blockSize){
     
    //declare pointers
    int *a, *b, *c, *dev_a, *dev_b, *dev_c;
    
    //HOST pinned memory allocation
    hipHostMalloc((void **)&a, N*sizeof(int));
    hipHostMalloc((void **)&b, N*sizeof(int));
    hipHostMalloc((void **)&c, N*sizeof(int));
    
    //GPU memory allocation
    hipMalloc(&dev_a, N * sizeof(int));
    hipMalloc(&dev_b, N * sizeof(int));
    hipMalloc(&dev_c, N * sizeof(int));
    
    //populate host arrays
    srand ( time(NULL) );
    for (int i = 0; i < N; i++) {
        a[i] = i; //a contains index number
        b[i] = rand() % 4; //b contains randoom values 0 to 3
    }
    
    //copy host arrays to GPU
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    
                          
    //performace measurement
    hipEvent_t kernel_start, kernel_stop;
    hipEventCreate(&kernel_start,0);
    hipEventCreate(&kernel_stop,0);
                          
    //select kernel to launch
    if(strcmp(op, "add")==0){
        hipEventRecord(kernel_start, 0);
        Add<<<blocks, threads>>>(dev_a, dev_b, dev_c);
        hipEventRecord(kernel_stop, 0);
    }
    
    else if(strcmp(op, "subtract")==0){
        hipEventRecord(kernel_start, 0);
        Subtract<<<blocks, threads>>>(dev_a, dev_b, dev_c);
        hipEventRecord(kernel_stop, 0);
    }
    
    else if(strcmp(op, "multiply")==0){
        hipEventRecord(kernel_start, 0);
        Mult<<<blocks, threads>>>(dev_a, dev_b, dev_c);
        hipEventRecord(kernel_stop, 0);
    }
    
    else if(strcmp(op, "mod")==0){
        hipEventRecord(kernel_start, 0);
        Mod<<<blocks, threads>>>(dev_a, dev_b, dev_c);
        hipEventRecord(kernel_stop, 0);
    }
    
    //synchronize 
    hipEventSynchronize(kernel_stop);
    hipDeviceSynchronize();
    
    
    //copy result back to host
    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
    
    //free allocated memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    
    //destroy events
    hipEventDestroy(kernel_start);
    hipEventDestroy(kernel_stop);
    
    
} 

int main(int argc, char** argv)
{
    // read command line arguments
    
    int totalThreads = 1024;
    int blockSize = 256;
    
    if (argc >= 3) {
        totalThreads = atoi(argv[2]);
    }
    if (argc >= 4) {
        blockSize = atoi(argv[3]);
    }

    int numBlocks = totalThreads/blockSize;

    // validate command line arguments
    if (totalThreads % blockSize != 0) {
        ++numBlocks;
        totalThreads = numBlocks*blockSize;
        
        printf("Warning: Total thread count is not evenly divisible by the block size\n");
        printf("The total number of threads will be rounded up to %d\n", totalThreads);
    
   
        }
    
    
    
    launchKernelVoid(argv[1], numBlocks, totalThreads, blockSize);
    
    launchKernelSharedStream(argv[1], numBlocks, totalThreads, blockSize);
    
    launchKernelShared(argv[1], numBlocks, totalThreads, blockSize);
}
