#include <stdio.h>
#include <hipblas.h>

const int N= 20;
const int MAX= 100;

//utility function prints arrays
void printVector(int * vec)
{   
    printf("{");
    for(int x = 0 ; x < N ; x++){
            printf(", %f", vec[x]);
        }
    printf("}\n");
}



void vectorSub(){
    //declare GPU pointers
    int *a, *b, *c, *dev_a, *dev_b, *dev_c;
    
    //HOST pinned memory allocation
    hipHostMalloc((int **)&a, N*sizeof(int));
    hipHostMalloc((int **)&b, N*sizeof(int));
    hipHostMalloc((int **)&c, N*sizeof(int));
    
    //GPU memory allocation
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));
    
    //populate host arrays
    srand ( time(NULL) );
    for (int i = 0; i < N; i++) {
        a[i] = rand() % MAX; //a rand between 0-99
        b[i] = rand() % MAX; //b rand between 0-99
    }
    
    //performace measurement
    hipEvent_t kernel_start, kernel_stop;
    hipEventCreate(&kernel_start,0);
    hipEventCreate(&kernel_stop,0);
    
    
    //initiallize
    hipEventRecord(kernel_start, 0);
    cublasInit();
    
    //set vector on device
    hipblasSetVector(N, sizeof(int), a, 1, dev_a, 1);
    hipblasSetVector(N, sizeof(int), b, 1, dev_b, 1);
       
    //saxpy with a=-1 to subtract a from b
    hipblasSaxpy(N, -1.0, dev_a, 1, dev_b, 1);
    
    //copy back and shutdown cublas
    hipblasGetVector(N, sizeof(int), dev_c, 1, c, 1);
    cublasShutdown();
    hipEventRecord(kernel_stop, 0);
    
    
    //output results and time
    float elapsedTime =0.0F;
    hipEventElapsedTime(&elapsedTime, kernel_start, kernel_stop);
    printf("Processed saxpy operations on vectors size %d using cublas in %f seconds using pinned memmory \n", N, elapsedTime);
    printVector(a);
    printVector(b);
    printVector(c);
    
    
    
    //free allocated memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    
    
    
    
}

int main(int argc, char** argv)
{
    vectorSub();
   
}
